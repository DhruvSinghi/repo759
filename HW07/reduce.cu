#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ int shared_mem[];
    float* sdata = shared_mem;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x
    if(i+threadIdx.x < n)
    sdata[threadIdx.x] = g_idata[i] + g_idata[i+threadIdx.x];

    for(unsigned int s = blockDim.x/2; s > 0; s>>=1)
    {
        if(threadIdx.x < s)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0) g_odata[blockIdx.x] = sdata[0];
}


__host__ void reduce(float **input, float **output, unsigned int N,unsigned int threads_per_block)
{
    while(N > 0)
    {
    unsigned int num_blocks = (N + threads_per_block - 1)/threads_per_block;

    reduce_kernel<<<num_blocks,threads_per_block>>>(*input,*output,N);

    hipDeviceSynchronize();

    *input = *output;

    N = num_blocks;
    }

}