#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include "matmul.cuh"

int main(int argc, char*argv[])
{
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = std::stoi(argv[1]);
    int block_dim = std::stoi(argv[2]);
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution <int> dist(-1.0,1.0);

    int*A = (int*)malloc(n*n*(sizeof(int)));
    int*B = (int*)malloc(n*n*(sizeof(int)));
    int*C = (int*)malloc(n*n*(sizeof(int)));
    int*d_A,*d_B,*d_C;

    for(int i = 0; i < n*n; i++)
    {
        A[i] = dist(generator);
        B[i] = dist(generator);
    }
    for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
        {
            std::cout<<A[i*n+j];
            if(j == n-1)
            {
                std::cout<<std::endl;
            }
        }
    }
    for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
        {
            std::cout<<B[i*n+j];
            if(j == n-1)
            {
                std::cout<<std::endl;
            }
        }
    }

    hipMalloc((void**)&d_A,sizeof(int) * n*n);
    hipMalloc((void**)&d_B,sizeof(int) * n*n);
    hipMalloc((void**)&d_C,sizeof(int) * n*n);

    hipMemcpy(d_A,A,sizeof(int)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,sizeof(int)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_C,C,sizeof(int)*n*n,hipMemcpyHostToDevice);

    hipEventRecord(start);
    matmul_1(d_A,d_B,d_C,n,block_dim);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipMemcpy(C,d_C,sizeof(int)*n*n,hipMemcpyDeviceToHost);
    hipEventElapsedTime(&ms, start, stop);

    for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
        {
            std::cout<<C[i*n+j];
            if(j == n-1)
            {
                std::cout<<std::endl;
            }
        }
    }
    std::cout<<C[(n-1)*n + n-1];
    std::cout<<std::endl; 
    std::cout<<"Time ELapsed "<<ms;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

}
