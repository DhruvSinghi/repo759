#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n,unsigned int block_dim)
{
    dim3 dimGrid((n+block_dim-1)/block_dim,(n+block_dim-1)/block_dim);
    dim3 dimBlock(block_dim,block_dim);
    matmul_1_kernel<<<(dimGrid,dimBlock,(2*block_dim*block_dim)*sizeof(int))>>>(A,B,C,n);
}

__global__ void matmul_1_kernel(int*A, int*B, int*C unsigned int n)
{
    extern __shared__ float shared_mem[];
    int* As = shared_mem;
    int* Bs = &shared_mem[blockDim.x*blockDim.x];
    int Csub = 0;
    for(int tile_idx = 0; tile_idx < n; tile_idx+=blockDim.x)
    {
        As[threadIdx.y*blockDim.x + threadIdx.x] = A[(blockIdx.y*blockDim.x+threadIdx.y)*n + (blockIdx.x*blockDim.x+threadIdx.x+tile_idx)];
        Bs[threadIdx.y*blockDim.x + threadIdx.x] = B[(blockIdx.y*blockDim.x+threadIdx.y+tile_idx)*n + (blockIdx.x*blockDim.x+threadIdx.x)];

        __syncthreads();

        for(int k = 0; k < blockDim.x; k++)
        {
            Csub += As[threadIdx.y*blockDim.x + k]*Bs[k*blockDim.x + threadIdx.x];
        }

    }

    __syncthreads();

    C[(blockIdx.y + threadIdx.y)*blockDim.x + threadIdx.x+blockIdx.x] = Csub;

}