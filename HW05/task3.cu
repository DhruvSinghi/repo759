#include <hip/hip_runtime.h>
#include <cstdio.h>
#include <random>
#include "vscale.cuh"

int main(int argc, char *argv[])
{
    const NUM_THREADS_PER_BLOCK 512
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int N = std::stoi(argv[1]);
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source()); 
    std::uniform_real_distribution<float> dist1(-10.0,10.0);
    std::uniform_real_distribution<float> dist2(0.0,1.0);
    
    float*a = (float*)malloc(N*N*(sizeof(float)));
    float*b = (float*)malloc(N*N*(sizeof(float)));

    for(int i = 0; i < N; i++)
    {
        a[i] = dist1(generator);
        b[i] = dist2(generator);
    }

    hipMalloc((void**)&d_a,sizeof(float) * N);
    hipMalloc((void**)&d_b,sizeof(float) * N);
    
    hipEventRecord(start);
    vscale<<<(N+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(d_a,d_b,N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    hipMemcpy(&b,d_b,sizeof(float)*N,hipMemcpyDeviceToHost);
    hipEventElapsedTime(&ms, start, stop);
    
    std::cout<<ms;
    std::cout << std::endl;
    std::cout<<b[0];
    std::cout << std::endl;
    std::cout<<b[N-1];
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    free(a);
    free(b);
    
}