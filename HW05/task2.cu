#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio.h>
#include <random>
#include <iostream>

__global__ void kernel(int *data,int a)
{
    data[threadIdx.x*blockIdx.x] = a*threadIdx.x + blockIdx.x;
}

int main()
{
    const int num_elements = 16;
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source()); 
    std::uniform_real_distribution<int> dist1(0,10);
    
    int a = dist1(generator);

    int hA[num_elements], *dA;

    hipMalloc((void**)&dA,sizeof(int) * num_elements);

    kernel<<<2,8>>>(dA,a);

    hipMemcpy(&hA,dA,sizeof(int)*num_elements,hipMemcpyDeviceToHost);

    for(int i = 0; i < num_elements; i++)
    {
        std::cout << hA[i] <<" ";
    }
    std::cout <<std::endl;
    hipFree(dA);
    
}