#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel()
{
    int b = 1;
    for(int i = threadIDx.x + 1; i >= 1 ; i--)
    {
       b = b*i;
    }
    printf("%d!=%d\n",threadIDx.x,b);
    
}
int main()
{
    kernel<<<1,8>>>();
    hipDeviceSynchronize();
    return 0;
}