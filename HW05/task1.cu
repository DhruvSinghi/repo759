#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel()
{
    int b = 1;
    for(int i = threadIdx.x + 1; i > 1; i--)
    {
	    b = b*i;
    }
    printf("%d!=%d\n",threadIdx.x+1,b);
    
}
int main()
{
    kernel<<<1,8>>>();
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
	    fprintf(stderr,"Kernel Launch Failed %s\n",hipGetErrorString(err));
    }
    return 0;
}
