#include "hip/hip_runtime.h"
#include "stencil.cuh"
#include <cstdio>
#include <cmath>

__host__ void stencil(const float* image,const float* mask, float* output, unsigned int n, unsigned int R, unsigned int threads_per_block)
    {
       stencil_kernel<<<(n+threads_per_block-1)/threads_per_block,threads_per_block>>>(image,mask,output,n,R);
    }

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R)
{
      __shared__ float image_s[blockDim.x];
      __shared__ float mask_s[2*R+1];
      __shared__ float output_s[blockDim.x];

      image_s[threadIdx.x] = image[blockDim.x*blockIdx.x+threadIdx.x];
      if(threadIdx.x < 2*R)
      mask_s[threadIdx.x] = mask[threadIdx.x];

      __syncthreads();

      int index = threadIdx.x;

      if(index < n)
      {
        output_s[index] = 0;
        for(int j = -R; j <= R; j++)
        {
            if(index + j >= 0 && index + j < n)
            output_s[index] += image_s[index + j]*mask_s[j + R];
            else
            output_s[index] += mask_s[j + R];
        }
      }

     // __syncthreads();

      output[index] = output_s[index];
}
