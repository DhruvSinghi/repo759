#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <cstdio>
#include <cmath>
__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n)
{
    int row = threadIdx.x + blockIdx.x*blockDim.x;
    int col = threadIdx.y + blockIdx.y*blockDim.y;
    float c = 0;
    if(row < n && col < n){
    for(int k =0; k < n; k++)
    {
        c += A[row*n + k]*B[k*n + col];
    }
    C[row*n+col] = c;
    }

}

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block)
{
    int block_size = sqrt(threads_per_block);
    dim3 threads_in_block (block_size,block_size);
    dim3 num_blocks ((n+block_size-1)/block_size,(n+block_size-1)/block_size);
    matmul_kernel<<<num_blocks,threads_in_block>>>(A,B,C,n);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
	    fprintf(stderr,"Kernel Launch Failed %s\n",hipGetErrorString(err));
    }
}
