#include "hip/hip_runtime.h"
#include "matmul.cuh"
__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    float c = 0;
    if(index < n)
    for(int k =0; k < n; k++)
    {
        c += A[index*n + k]*B[k*n + index];
    }
    C[index*n+index] = c;

}

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block)
{
    matmul_kernel<<<(n+threads_per_block-1)/threads_per_block,threads_per_block>>>(A,B,C,n);
}