#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include "matmul.cuh"

int main(int argc, char*argv[])
{
    const int NUM_THREADS_PER_BLOCK = 256;
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = std::stoi(argv[1]);
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution<float> dist(-1.0,1.0);

    float*a = (float*)malloc(n*n*(sizeof(float)));
    float*b = (float*)malloc(n*n*(sizeof(float)));
    float*c = (float*)malloc(n*n*(sizeof(float)));
    float*d_a,*d_b,*d_c;

    for(int i = 0; i < n*n; i++)
    {
        a[i] = dist(generator);
        b[i] = dist(generator);
    }
   /* for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
        {
            std::cout<<a[i*n+j];
            if(j == n-1)
            {
                std::cout<<std::endl;
            }
        }
    }*/
    /*for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
        {
            std::cout<<b[i*n+j];
            if(j == n-1)
            {
                std::cout<<std::endl;
            }
        }
    }*/

    hipMalloc((void**)&d_a,sizeof(float) * n*n);
    hipMalloc((void**)&d_b,sizeof(float) * n*n);
    hipMalloc((void**)&d_c,sizeof(float) * n*n);

    hipMemcpy(d_a,a,sizeof(float)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,sizeof(float)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_c,c,sizeof(float)*n*n,hipMemcpyHostToDevice);

    hipEventRecord(start);
    matmul(d_a,d_b,d_c,n,NUM_THREADS_PER_BLOCK);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipMemcpy(c,d_c,sizeof(float)*n*n,hipMemcpyDeviceToHost);
    hipEventElapsedTime(&ms, start, stop);

   /* for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
        {
            std::cout<<c[i*n+j];
            if(j == n-1)
            {
                std::cout<<std::endl;
            }
        }
    }*/
    std::cout<<c[(n-1)*n + n-1];
    std::cout<<std::endl; 
    std::cout<<"Time ELapsed "<<ms;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

}
