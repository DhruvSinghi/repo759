#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include "stencil.cuh"

int main(int argc, char*argv[])
{
    int NUM_THREADS_PER_BLOCK = 1024;
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = std::stoi(argv[1]);
    int R = std::stoi(argv[2]);
    int NUM_THREADS_PER_BLOCK = std::stoi(argv[3]);
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution<float> dist(-1.0,1.0);

    float*image = (float*)malloc(n*(sizeof(float)));
    float*mask = (float*)malloc(R*(sizeof(float)));
    float*output = (float*)malloc(n*(sizeof(float)));
    float*d_image,*d_mask,*d_output;

    for(int i = 0; i < n; i++)
    {
        image[i] = dist(generator);
    }
    for(int i = 0; i < R; i++)
    {
        mask[i] = dist(generator);
    }
    for(int i = 0; i < n; i++)
    {
        std::cout<<image[i]<<" ";
    }
    for(int i = 0; i < n; i++)
    {
        std::cout<<mask[i]<<" ";
    }

    hipMalloc((void**)&d_image,sizeof(float)*n);
    hipMalloc((void**)&d_output,sizeof(float)*n);
    hipMalloc((void**)&d_mask,sizeof(float)*R);

    hipMemcpy(d_image,image,sizeof(float)*n,hipMemcpyHostToDevice);
    hipMemcpy(d_mask,mask,sizeof(float)*R,hipMemcpyHostToDevice);
    hipMemcpy(d_output,output,sizeof(float)*n,hipMemcpyHostToDevice);

    hipEventRecord(start);
    stencil(d_image,d_mask,d_output,n,R,NUM_THREADS_PER_BLOCK);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipMemcpy(output,d_output,sizeof(float)*n,hipMemcpyDeviceToHost);
    hipEventElapsedTime(&ms, start, stop);

    for(int i = 0; i < n; i++)
    {
        std::cout<<output[i]<<" ";
    }
    std::cout<<std::endl; 
    std::cout<<"Time ELapsed "<<ms;

    hipFree(d_image);
    hipFree(d_output);
    hipFree(d_mask);
    free(image);
    free(output);
    free(mask);

}
